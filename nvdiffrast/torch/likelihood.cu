
#include <hip/hip_runtime.h>
inline __device__ int index(const int i, const int j, const int k, int width, int height, int depth) {
     return k * (width * height * 4) + i * (width * 4) + j * 4;
}

__global__ void threedp3_likelihood(float *pos, float time, int width, int height, int depth)
{   
    unsigned int i = blockIdx.x;
    unsigned int j = blockIdx.y;
    unsigned int k = threadIdx.x;

    int idx = index(i,j,k, width, height, depth);

    // write output vertex
    pos[idx] = pos[idx];
}
